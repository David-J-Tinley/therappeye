#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

const int WWIDTH = 1920;
const int WHEIGHT = 1080;

__global__ void drawCircle(unsigned char* pixels,
                            int width,
                            int height,
                            int centerX,
                            int centerY,
                            int radius,
                            unsigned char r,
                            unsigned char g,
                            unsigned char b)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int dx = x - centerX;
    int dy = y - centerY;
    int distanceSquared = dx * dx + dy * dy;

    if (distanceSquared <= radius * radius)
    {
        int index = (y * width + x) * 4;
        pixels[index] = r;
        pixels[index + 1] = g;
        pixels[index + 2] = b;
        pixels[index + 3] = 255;
    }
}

int main()
{
    sf::RenderWindow window(sf::VideoMode(WWIDTH, WHEIGHT), "Vision Therapy");
    window.setFramerateLimit(60);

    sf::Texture texture;
    texture.create(WWIDTH, WHEIGHT);
    sf::Sprite sprite(texture);

    unsigned char* pixels = new unsigned char[WWIDTH * WHEIGHT * 4];
    hipMallocManaged(&pixels, WWIDTH * WHEIGHT * 4);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(WWIDTH / threadsPerBlock.x, WHEIGHT / threadsPerBlock.y);

    
    int radius = 50;
    int speed = 10;

    int x = radius;
    int y = (WHEIGHT / 2) - radius;

    while (window.isOpen()) {
        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed) {
                window.close();
                break;
            }
            if (event.type == sf::Event::KeyPressed) {
                if (event.key.code == sf::Keyboard::Q) {
                    window.close();
                    break;
                }
            }
        }

        x += speed;
        if (x + radius > WWIDTH) {
            x = WWIDTH - radius;
            speed = -speed;
        } else if (x - radius < 0) {
            x = radius;
            speed = -speed;
        }

        // Clear the pixel array
        hipMemset(pixels, 0, WWIDTH * WHEIGHT * 4);

        drawCircle<<<numBlocks, threadsPerBlock>>>(pixels,
                                                    WWIDTH, WHEIGHT,
                                                    x,
                                                    y,
                                                    radius,
                                                    0, 255, 0);
        
        hipDeviceSynchronize();

        texture.update(pixels, WWIDTH, WHEIGHT, 0, 0);
        window.clear(sf::Color::Yellow);
        window.draw(sprite);
        window.display();
    }

    hipFree(pixels);

    return 0;
}